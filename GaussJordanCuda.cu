
#include <hip/hip_runtime.h>
#include <iostream> 
#include <fstream>
#include <cstdlib>
#include <cmath>
#define EPS 1e-6
#include <ctime>

#define N 500
#define THREADS 32

using namespace std;


typedef float dato;

void printmatrice(dato *m){
  for(int i=0; i<N; i++){
    cout << endl;
    for(int j=0; j<N+1; j++){
      cout <<"\t"<< m[i*(N+1)+j];
    }
   }

  cout << endl;
}

void confrontaSol(dato sol[N],dato solmath[N]){
  cout<<"Soluzioni con differenza > 0.0001 rispetto al risultato esatto: "<<endl;
  for(int i=0; i<N; i++){
	if(abs(sol[i]/solmath[i]-1)>0.0001)
	cout<<i<<" "<< abs(sol[i]/solmath[i]-1)<<endl;
  }
}

__global__ void completa_matrice(dato *a) {

	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int offset = gridDim.x*blockDim.x*i + j;
	
	if(i >= N || j >= N+1) 
		a[offset] = 0.;


};



__global__ void triangolo(dato* matric_dev, int index){
	
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int offset = gridDim.x*blockDim.x*i + j;

	int row = index*gridDim.x*blockDim.x;
	int ind = row+j;
	//matric_dev[threadIdx.x]=2;

	
	if(i>index && j>index){
		if(matric_dev[row+index]!=0){
			dato coef = matric_dev[i*gridDim.x*blockDim.x+index]/matric_dev[row+index];

			matric_dev[offset] -= matric_dev[ind]*coef;
			//matric_dev[offset]=4;
		}
	}
}



__global__ void findmax (dato* matrice_dev, dato* pivot_dev, int col, int *Max){
	
	__shared__ dato cachecontrol [THREADS];
	__shared__ int indices [THREADS];
	
	int row = blockIdx.x*blockDim.x + threadIdx.x;
	indices[threadIdx.x]=row;

	if(row>=col){
		cachecontrol[threadIdx.x] = abs(matrice_dev[(row)*(gridDim.x*blockDim.x)+col]);
	} else {
		cachecontrol[threadIdx.x] = 0;
	}
	
	__syncthreads();

	int i=THREADS/2;
	if(threadIdx.x<i){
		while (i!=0){
			if(threadIdx.x<i){
				if(cachecontrol[threadIdx.x]<cachecontrol[threadIdx.x+i]){
					cachecontrol[threadIdx.x] = cachecontrol[threadIdx.x+i];
					indices[threadIdx.x]=indices[threadIdx.x+i];
				}
				i/=2;
				__syncthreads();
			} else {
				i=0;
				}
		}
	}

	if(threadIdx.x==0){
		pivot_dev[blockIdx.x]=cachecontrol[0];
		Max[blockIdx.x]=indices[0];
	}

	__syncthreads();

	if(threadIdx.x==0 && blockIdx.x==0){
		for( int i=0; i<gridDim.x; i++){
			if(pivot_dev[0]<pivot_dev[i]){
				pivot_dev[0]=pivot_dev[i];
				Max[0]=Max[i];
			}
		}
	}
}

__global__ void pivoting (dato* matrice_dev, int col, int* rowmax ){ // metti int rowmax
	
	
	int lenrow = gridDim.x*blockDim.x;
	int row = blockIdx.x*blockDim.x + threadIdx.x;

	dato appo = matrice_dev[lenrow*rowmax[0]+row];
	matrice_dev[lenrow*rowmax[0]+row] = matrice_dev[lenrow*col+row];
	matrice_dev[lenrow*col+row]=appo;
}


void solve(dato *m, dato sol[]){
    sol[N-1]=m[N*(N+1)-1]/m[N*(N+1)-2];
    for(int i=N-2; i>=0; i--){
      for(int j=N-1; j>i; j--){
	m[i*(N+1)+N] -= m[i*(N+1)+j]*sol[j];
      }
      if(m[i*(N+1)+i]!=0){
        sol[i]=m[i*(N+1)+N]/m[i*(N+1)+i];
      }
      else{
        sol[i]=0;
      }
   }
}


void triangolizza( dato *matrice ){
	
	dato* matrice_dev; 
	dato *pivot_dev;
	int *Max;
	//int *Maxcpu;
	int nblock1 = N/THREADS + 1; // y, colonne (j)
	int nblock2 = (N+1)/THREADS +1; // x, righe (i)
	int width1= nblock1*THREADS;
	int width2= nblock2*THREADS;
	
	hipMalloc((void**)&matrice_dev,width1*width2*sizeof(dato));
	hipMalloc((void**)&pivot_dev,nblock1*sizeof(dato));
	hipMalloc((void**)&Max,nblock1*sizeof(int));
	//Maxcpu = new int [nblock1];
	
	cout<< "numero blocchi: "<<nblock2<<" x "<< nblock1 <<" width: "<<width2<<" x "<<width1<<endl;

	hipMemcpy2D(matrice_dev,width2*sizeof(dato),matrice,(N+1)*sizeof(dato),(N+1)*sizeof(dato),N,hipMemcpyHostToDevice);

	dim3 c_threads(THREADS,THREADS);
	dim3 c_blocks(nblock2,nblock1);

	completa_matrice<<<c_blocks,c_threads>>>(matrice_dev);

	for( int i=0; i<N-1; i++){
		findmax<<<nblock1,THREADS>>>(matrice_dev, pivot_dev, i, Max);
		//cudaMemcpy(Maxcpu,Max,nblock1*sizeof(int),cudaMemcpyDeviceToHost);
		//cout<<Maxcpu[0]<<endl;
		pivoting<<<nblock1,THREADS>>>(matrice_dev, i, Max);
		triangolo<<<c_blocks,c_threads>>>(matrice_dev,i);
	}
	
	hipMemcpy2D(matrice,(N+1)*sizeof(dato),matrice_dev,width2*sizeof(dato),(N+1)*sizeof(dato),N,hipMemcpyDeviceToHost);

	hipFree(pivot_dev);
	hipFree(matrice_dev);
  }




int main (){

  hipSetDevice(0);	
	
  //clock_t t;
  int dim;
  dato *matrice;
  dato soluzioni [N], soluzionimath[N];

  ifstream GetMatrix;
  ifstream GetTerm;
  ifstream GetSol;

  matrice=new dato [N*(N+1)];

  //t=clock();

  GetMatrix.open("matrix.txt");
  if(GetMatrix.fail()){
     cout<< endl << "Problema apertura file di ingresso dati! Esco!";
     return 0;
  }

  GetTerm.open("term.txt");
  if(GetTerm.fail()){
     cout<< endl << "Problema apertura file di ingresso dati! Esco!";
     return 0;
  }

  GetMatrix>>dim;

  for(int i=0; i<N; i++){
    GetTerm>>matrice[i*(N+1)+N];
    for(int j=0; j<N; j++){
      GetMatrix>>matrice[i*(N+1)+j];
    }
  }

  GetSol.open("solutions.txt");
  if(GetSol.fail()){
     cout<< endl << "Problema apertura file di ingresso dati! Esco!";
     return 0;
  }


  for(int i=0; i<N; i++){
	GetSol>> soluzionimath[i];
  }
  

  GetMatrix.close();
  GetTerm.close();


  //printmatrice(matrice);
  triangolizza(matrice);	
  //printmatrice(matrice);
  solve(matrice, soluzioni);

  cout<< "Soluzioni CUDA: "<<endl;
  for(int i=0; i<N; i++){
	cout<<soluzioni[i]<<endl;
  }
  
  cout<< "Soluzioni mathematica: "<<endl;
  for(int i=0; i<N; i++){
	cout<<soluzionimath[i]<<endl;
  }
  
  confrontaSol(soluzioni, soluzionimath);
	
  //cudaEvent_t start,stop;
  //cudaEventCreate(&start);
  //cudaEventCreate(&stop);
  //cudaEventRecord(start,0);

  
  return 0;
}


