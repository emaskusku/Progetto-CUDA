
#include <hip/hip_runtime.h>
#include <iostream> 
#include <fstream>
#include <cstdlib>
#include <cmath>
#define EPS 1e-6
#include <ctime>

#define N 4
#define THREADS 32

using namespace std;


typedef float dato;

void printmatrice(float *m){
  for(int i=0; i<N; i++){
    cout << endl;
    for(int j=0; j<N+1; j++){
      cout <<"\t"<< m[i*(N+1)+j];
    }
   }

  cout << endl;
}

__global__ void triangola (dato *a_dev){

	__shared__ float coef [THREADS];
	int ncicli= (N+1-blockIdx.x)/THREADS + 1;

	if(blockIdx.x==1){
		//a_dev[3]=100000;
	}
    
       for(int i=blockIdx.x+1; i<N ; i++){
	   if(a_dev[blockIdx.x*(N+1)+blockIdx.x]!=0){
	   for(int j=0; j<ncicli; j++){
	     int index = threadIdx.x + j*THREADS + (N+1)*blockIdx.x+blockIdx.x; //dovrebbe andare bene
	      if(index < N+1){
		coef[threadIdx.x]=a_dev[index]/a_dev[blockIdx.x*(N+1)+blockIdx.x];
		a_dev[i*(N+1)+threadIdx.x+j*THREADS+blockIdx.x]-= coef[threadIdx.x]*a_dev[i*(N+1)+threadIdx.x+j*THREADS+blockIdx.x];
	  }
	}
       }
     }
     __syncthreads();
    }

int main (){

  hipSetDevice(0);	
	
  clock_t t;
  int dim;
  float *matrice;
  float termini[N]; //termini noti
  float soluzioni [N],soluzionimath[N];
  ifstream GetMatrix;
  ifstream GetTerm;
  ifstream GetSol;
  dato *a_dev;	
  int nblock = (N+THREADS-1)/THREADS;	
  int width = nblock*THREADS;
  size_t size = N*(N+1)*sizeof(dato);
  hipMalloc((void**)&a_dev,size);

  matrice=new float [N*(N+1)];

  t=clock();

  GetMatrix.open("matrix.txt");
  if(GetMatrix.fail()){
     cout<< endl << "Problema apertura file di ingresso dati! Esco!";
     return 0;
  }

  GetTerm.open("term.txt");
  if(GetTerm.fail()){
     cout<< endl << "Problema apertura file di ingresso dati! Esco!";
     return 0;
  }

  GetMatrix>>dim;

  for(int i=0; i<N; i++){
    GetTerm>>matrice[i*(N+1)+N];
    for(int j=0; j<N; j++){
      GetMatrix>>matrice[i*(N+1)+j];
    }
  }

  GetMatrix.close();
  GetTerm.close();


  printmatrice(matrice);

  hipMemcpy(a_dev, matrice, size, hipMemcpyHostToDevice);
  triangola<<<(dim-1),THREADS>>>(a_dev);
  hipMemcpy(matrice, a_dev, size, hipMemcpyDeviceToHost);

  printmatrice(matrice);
	
  //cudaEvent_t start,stop;
  //cudaEventCreate(&start);
  //cudaEventCreate(&stop);
  //cudaEventRecord(start,0);

  


  hipFree(a_dev);
  return 0;
}


