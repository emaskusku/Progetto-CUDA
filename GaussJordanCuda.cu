
#include <hip/hip_runtime.h>
#include <iostream> 
#include <fstream>
#include <cstdlib>
#include <cmath>
#define EPS 1e-6
#include <ctime>

#define N 7000
#define THREADS 32
#define THREADS1 1024

using namespace std;


typedef double dato;

void printmatrice(dato *m){
  for(int i=0; i<N; i++){
    cout << endl;
    for(int j=0; j<N+1; j++){
      cout <<"\t"<< m[i*(N+1)+j];
    }
   }

  cout << endl;
}

void confrontaSol(dato *sol,dato *solmath){
  cout<<"Soluzioni con differenza > 0.00001 rispetto al risultato esatto: "<<endl;
  for(int i=0; i<N; i++){
	if(abs(sol[i]/solmath[i]-1)>0.00001)
	cout<<i<<" "<< abs(sol[i]/solmath[i]-1)<<endl;
  }
}

__global__ void completa_matrice(dato *a) {

	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int offset = gridDim.x*blockDim.x*i + j;
	
	if(i >= N || j >= N+1) 
		a[offset] = 0.;


};


__global__ void triangolo(dato* matric_dev, int index){
	

	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int offset = gridDim.x*blockDim.x*i + j;

	int row = index*gridDim.x*blockDim.x;
	int ind = row+j;
	//matric_dev[threadIdx.x]=2;

	
	if(i>index && j>index){
		if(matric_dev[row+index]!=0){
			dato coef = matric_dev[i*gridDim.x*blockDim.x+index]/matric_dev[row+index];

			matric_dev[offset] -= matric_dev[ind]*coef;
			//matric_dev[offset]=4;
		}
	}
	__syncthreads();
}



__global__ void findmax (dato* matrice_dev, dato* pivot_dev, int col, int *Max){
	
	int row = blockIdx.x*blockDim.x + threadIdx.x;
	int len = int((N+1)/THREADS+1)*THREADS;

	__shared__ dato cachecontrol [THREADS1];
	__shared__ int indices [THREADS1];
	
	indices[threadIdx.x]=row;

	if(row>=col && row < len){
	cachecontrol[threadIdx.x] = abs(matrice_dev[(row)*(len)+col]);
	} else {
		cachecontrol[threadIdx.x] = 0;
	}
	
	__syncthreads();

	int i=blockDim.x/2;

	while (i!=0){
		if(threadIdx.x<i){
		if(cachecontrol[threadIdx.x]<cachecontrol[threadIdx.x+i]){
			cachecontrol[threadIdx.x] = cachecontrol[threadIdx.x+i];
			indices[threadIdx.x]=indices[threadIdx.x+i];
		}
		}

		i/=2;
		__syncthreads();

	}

	if(threadIdx.x==0){
		pivot_dev[blockIdx.x]=cachecontrol[0];
		Max[blockIdx.x]=indices[0];
	}

	__syncthreads();
}


__global__ void getmax(dato * pivot_dev, int * Max, int nblock3){

	if(threadIdx.x==0 && blockIdx.x==0){
		for( int i=0; i<nblock3; i++){
			if(pivot_dev[0]<pivot_dev[i]){
				pivot_dev[0]=pivot_dev[i];
				Max[0]=Max[i];
			}
		}
	}
	__syncthreads();
}


__global__ void findmax1(dato *matrice_dev, dato* pivrow, int col, int *Maxind){ // da mandare con 1 blocco e THREADS*nblock threads

	Maxind[threadIdx.x]=threadIdx.x;

	if(threadIdx.x>=col){
		pivrow[threadIdx.x] = matrice_dev[threadIdx.x*blockDim.x+col];
	} else {
		pivrow[threadIdx.x] = 0;
	}
	
	__syncthreads();

	int i=blockDim.x/2;

	while (i!=0){
		if(threadIdx.x<i){
		if(pivrow[threadIdx.x] < pivrow[threadIdx.x+i]){
			pivrow[threadIdx.x] = pivrow[threadIdx.x+i];
			Maxind[threadIdx.x] = Maxind[threadIdx.x+i];
		}
		}

		i/=2;
		__syncthreads();

	}
}


__global__ void pivoting (dato* matrice_dev, dato* pivot_dev, int col, int* Max ){

	if(pivot_dev[0]==0){
		__syncthreads();
		return;
	}

	int lenrow = int((N+1)/THREADS+1)*THREADS;
	int row = blockIdx.x*blockDim.x + threadIdx.x;

	if(row < int((N+1)/THREADS+1)*THREADS){
		dato appo = matrice_dev[lenrow*Max[0]+row];
		matrice_dev[lenrow*Max[0]+row] = matrice_dev[lenrow*col+row];
		matrice_dev[lenrow*col+row]=appo;
		//Max[blockIdx.x]=0;
		//pivot_dev[blockIdx.x]=0;
	}
	__syncthreads();
}



void solve(dato *m, dato sol[]){
    sol[N-1]=m[N*(N+1)-1]/m[N*(N+1)-2];
    for(int i=N-2; i>=0; i--){
      for(int j=N-1; j>i; j--){
	m[i*(N+1)+N] -= m[i*(N+1)+j]*sol[j];
      }
      if(m[i*(N+1)+i]!=0){
        sol[i]=m[i*(N+1)+N]/m[i*(N+1)+i];
      }
      else{
        sol[i]=0;
      }
   }
}


__global__ void solveCuda(dato*m, dato *sol, int block1){

    int len = block1*THREADS;
    sol[N-1]=m[(N-1)*(len)+N]/m[(N-1)*(len)+N-1];

    for(int i=N-2; i>=0; i--){
      for(int j=N-1; j>i; j--){
	m[i*(len)+N] -= m[i*(len)+j]*sol[j];
      }
      if(m[i*(len)+i]!=0){
        sol[i]=m[i*(len)+N]/m[i*(len)+i];
      }
      else{
        sol[i]=0;
      }
   }

   
}


/*
__global__ void triangolocuda(int nblock1, dato* matrice_dev, dato* pivot_dev, int* Max){

	dim3 c_threads(THREADS,THREADS);
	dim3 c_blocks(nblock2,nblock1);
	
	for( int i=0; i<N-1; i++){
		findmax<<<nblock1,THREADS>>>(matrice_dev, pivot_dev, i, Max);
		getmax<<<1,THREADS>>>(pivot_dev, Max, nblock1);
		pivoting<<<nblock1,THREADS>>>(matrice_dev, pivot_dev, i, Max);
		triangolo<<<c_blocks,c_threads>>>(matrice_dev,i);
		cudaDeviceSynchronize();
	}
}
*/

dato* triangolizza( dato *matrice ){
	
	dato* matrice_dev; 
	dato *pivot_dev;
	dato* solcuda;
	dato *solcpu;
	int *Max;
	int nblock1 = N/THREADS + 1; // y, colonne (j)
	int nblock2 = (N+1)/THREADS + 1; // x, righe (i)
	int nblock3 = (N+1)/(THREADS1) + 1;
	nblock1=nblock2;
	int width1= nblock1*THREADS;
	int width2= nblock2*THREADS;
	
	solcpu = new dato [N];
	hipMalloc((void**)&matrice_dev,width2*width1*sizeof(dato));
	hipMalloc((void**)&pivot_dev,nblock3*sizeof(dato));
	hipMalloc((void**)&Max,nblock3*sizeof(int));
	hipMalloc((void**)&solcuda,N*sizeof(dato));

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	cout<< "numero blocchi kernel 2D: "<<nblock2<<" x "<< nblock1 <<", width: "<<width2<<" x "<<width1<<endl;

	hipMemcpy2D(matrice_dev,width2*sizeof(dato),matrice,(N+1)*sizeof(dato),(N+1)*sizeof(dato),N,hipMemcpyHostToDevice);


	dim3 c_threads(THREADS,THREADS);
	dim3 c_blocks(nblock2,nblock1);

	completa_matrice<<<c_blocks,c_threads>>>(matrice_dev);

	//triangolocuda<<<1,1>>>(nblock1, matrice_dev, pivot_dev, Max);
	cout<< "numero blocchi kernel 1D: "<<nblock3<<", width: "<<THREADS1*nblock3<<endl;

	for( int i=0; i<N-1; i++){

		findmax<<<nblock3,THREADS1>>>(matrice_dev, pivot_dev, i, Max);
		//cudaDeviceSynchronize();
		getmax<<<1,1>>>(pivot_dev, Max, nblock3);
		//cudaDeviceSynchronize();
		pivoting<<<nblock3,THREADS1>>>(matrice_dev, pivot_dev, i, Max);
		//cudaDeviceSynchronize();
		triangolo<<<c_blocks,c_threads>>>(matrice_dev,i);
		//cudaDeviceSynchronize();
	}

	solveCuda<<<1,1>>>(matrice_dev, solcuda, nblock1);

	hipMemcpy(solcpu, solcuda, N*sizeof(dato), hipMemcpyDeviceToHost);
	//cudaMemcpy2D(matrice,(N+1)*sizeof(dato),matrice_dev,width2*sizeof(dato),(N+1)*sizeof(dato),N,cudaMemcpyDeviceToHost);


	//printmatrice(matrice);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsed;
	hipEventElapsedTime(&elapsed,start,stop);
	cout<<"tempo: " << elapsed/1000. << endl;
	
	//dealloco 
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(solcuda);
	hipFree(pivot_dev);
	hipFree(matrice_dev);
	return solcpu;
  }




int main (){

  hipSetDevice(2);	
	
  //clock_t t;
  int dim;
  dato *matrice;
  //dato soluzioni [N];
  dato soluzionimath[N];
  dato* solfromcuda;
  ifstream GetMatrix;
  ifstream GetTerm;
  ifstream GetSol;

  matrice=new dato [N*(N+1)];

  //t=clock();

  GetMatrix.open("matrix.txt");
  if(GetMatrix.fail()){
     cout<< endl << "Problema apertura file di ingresso dati! Esco!";
     return 0;
  }

  GetTerm.open("term.txt");
  if(GetTerm.fail()){
     cout<< endl << "Problema apertura file di ingresso dati! Esco!";
     return 0;
  }

  GetMatrix>>dim;

  for(int i=0; i<N; i++){
    GetTerm>>matrice[i*(N+1)+N];
    for(int j=0; j<N; j++){
      GetMatrix>>matrice[i*(N+1)+j];
    }
  }

  GetSol.open("solutions.txt"); //occhio da quale file prendi
  if(GetSol.fail()){
     cout<< endl << "Problema apertura file di ingresso dati! Esco!";
     return 0;
  }


  for(int i=0; i<N; i++){
	GetSol>> soluzionimath[i];
  }
  

  GetMatrix.close();
  GetTerm.close();


  //printmatrice(matrice);
  solfromcuda = triangolizza(matrice);	
  //printmatrice(matrice);
  //solve(matrice, soluzioni);

/*
  cout<< "Soluzioni CUDA: "<<endl;
  for(int i=0; i<N; i++){
	cout<<soluzioni[i]<<endl;
  }
  
  cout<< "Soluzioni mathematica: "<<endl;
  for(int i=0; i<N; i++){
	cout<<soluzionimath[i]<<endl;
  }
 */

  cout<<solfromcuda[0]<<" "<<soluzionimath[0]<<endl;
  cout<<solfromcuda[1]<<" "<<soluzionimath[1]<<endl;
  cout<<solfromcuda[2]<<" "<<soluzionimath[2]<<endl;
  cout<<solfromcuda[3]<<" "<<soluzionimath[3]<<endl; 
  cout<<solfromcuda[4]<<" "<<soluzionimath[4]<<endl;

/*
  cout<<soluzioni[0]<<" "<<soluzionimath[0]<<endl;
  cout<<soluzioni[1]<<" "<<soluzionimath[1]<<endl;
  cout<<soluzioni[2]<<" "<<soluzionimath[2]<<endl;
  cout<<soluzioni[3]<<" "<<soluzionimath[3]<<endl; 
  cout<<soluzioni[4]<<" "<<soluzionimath[4]<<endl;
*/
  //confrontaSol(soluzioni, soluzionimath);
  confrontaSol(solfromcuda, soluzionimath);
	
  //cudaEvent_t start,stop;
  //cudaEventCreate(&start);
  //cudaEventCreate(&stop);
  //cudaEventRecord(start,0);

  
  return 0;
}

